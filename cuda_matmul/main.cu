#include "hip/hip_runtime.h"
#include <cstdio>

#include <hip/hip_runtime.h>

#include "main.cuh"

#define gpuCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

namespace {

constexpr size_t tile = 16;
__global__
void matmulV1(const float* a, const float* b, float* c, int i, int j, int k) {
  // Figure out the output element I am writing to
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row > i || col > k) {
    return;
  }

  float dotp = 0.0;
  for (auto idx = 0U ; idx < j ; ++idx) {
    dotp += (a[row * j + idx] * b[col + idx * j]);
  }

  c[row * k + col] = dotp;
  
}

__global__
void matmulV2(const float* a, const float* b, float* c, int i, int j, int k) {
  __shared__ float rowa[tile][tile];
  __shared__ float cola[tile][tile];
  
  // Figure out the output element I am writing to
  // For this to work blockDim.x == tile, blockDim.y = tile
  const int col = blockIdx.x * tile + threadIdx.x;
  const int row = blockIdx.y * tile + threadIdx.y;
  if (row > i || col > k) {
    return;
  }

  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  float dotp = 0.0;  
  for (auto idx = 0 ; idx < j ; idx += tile) {
    if ((idx + tx) < j)
      rowa[ty][tx] = a[row * j + idx + tx];
    if ((idx + ty < j))
      cola[ty][tx] = b[col + (idx + ty) * j];
    __syncthreads();


    for (auto i = 0U ; i < tile ; ++i) {
      // Don't add elements beyond the vector boundaries
      dotp += rowa[ty][i] * cola[i][tx];
    }
    __syncthreads();    
    
  }

  c[row * k + col] = dotp;
  
}

template <typename T>
void kernelRunner(const std::vector<float>& a,
              const std::vector<float>& b,
              std::vector<float>& c,              
              int i,
              int j,
            int k,
            T fn) {
  const auto a_n = a.size() * sizeof(float);
  const auto b_n = b.size() * sizeof(float);
  const auto c_n = c.size() * sizeof(float);
  float* d_a;
  float* d_b;
  float* d_c;
  gpuCheck(hipMalloc(&d_a, a_n));
  gpuCheck(hipMalloc(&d_b, b_n));
  gpuCheck(hipMalloc(&d_c, c_n));

  gpuCheck(hipMemcpy(d_a, a.data(), a_n, hipMemcpyHostToDevice));
  gpuCheck(hipMemcpy(d_b, b.data(), b_n, hipMemcpyHostToDevice));
  gpuCheck(hipMemcpy(d_c, c.data(), c_n, hipMemcpyHostToDevice));  

  dim3 dimGrid(ceil(i/(float)tile), ceil(k/(float)tile), 1);
  dim3 dimBlock(tile, tile, 1);

  printf("GridDim %d, %d, %d\n", dimGrid.x, dimGrid.y, dimGrid.z);
  printf("BlockDim %d, %d, %d\n", dimBlock.x, dimBlock.y, dimBlock.z);  

  for (auto i = 0U ; i < 200 ; ++i) {
    fn<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, i, j, k);
    gpuCheck(hipPeekAtLastError());
    hipDeviceSynchronize();
  }

  gpuCheck(hipMemcpy(c.data(), d_c, c_n, hipMemcpyDeviceToHost));
  gpuCheck(hipFree(d_a));
  gpuCheck(hipFree(d_b));
  gpuCheck(hipFree(d_c));  
}

} // unnamed namespace

namespace wrapper {

void matMulV1(const std::vector<float>& a,
              const std::vector<float>& b,
              std::vector<float>& c,              
              int i,
              int j,
              int k) {
  kernelRunner(a, b, c, i, j, k, ::matmulV1);
}

// TODO: hook it up with the kernel that uses shared memory
void matMulV2(const std::vector<float>& a,
              const std::vector<float>& b,
              std::vector<float>& c,              
              int i,
              int j,
              int k) {
  kernelRunner(a, b, c, i, j, k, ::matmulV2);
}

void print_cuda_properties() {
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  fprintf(stdout, "Warp size: %d\n", props.warpSize);
  fprintf(stdout, "Max grid size: %d, %d, %d\n", props.maxGridSize[0], props.maxGridSize[1], props.maxGridSize[2]);  
}
}
