#include "hip/hip_runtime.h"
#include <cstdio>

#include <hip/hip_runtime.h>

#include "main.cuh"

#define gpuCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

namespace {

__global__
void matmulV1(const float* a, const float* b, float* c, int i, int j, int k) {
  // Figure out the output element I am writing to
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row > i || col > k) {
    return;
  }

  float dotp = 0.0;
  for (auto idx = 0U ; idx < j ; ++idx) {
    dotp += (a[row * j + idx] * b[col + idx * j]);
  }

  c[row * k + col] = dotp;
  
}

template <typename T>
void kernelRunner(const std::vector<float>& a,
              const std::vector<float>& b,
              std::vector<float>& c,              
              int i,
              int j,
            int k,
            T fn) {
  const auto a_n = a.size() * sizeof(float);
  const auto b_n = b.size() * sizeof(float);
  const auto c_n = c.size() * sizeof(float);
  float* d_a;
  float* d_b;
  float* d_c;
  gpuCheck(hipMalloc(&d_a, a_n));
  gpuCheck(hipMalloc(&d_b, b_n));
  gpuCheck(hipMalloc(&d_c, c_n));

  gpuCheck(hipMemcpy(d_a, a.data(), a_n, hipMemcpyHostToDevice));
  gpuCheck(hipMemcpy(d_b, b.data(), b_n, hipMemcpyHostToDevice));
  gpuCheck(hipMemcpy(d_c, c.data(), c_n, hipMemcpyHostToDevice));  

  dim3 dimGrid(ceil(i/4.0), ceil(k/4.0), 1);
  dim3 dimBlock(4, 4, 1);

  printf("Grid %d, %d, %d\n", dimGrid.x, dimGrid.y, dimGrid.z);
  
  fn<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, i, j, k);
  hipDeviceSynchronize();

  gpuCheck(hipMemcpy(c.data(), d_c, c_n, hipMemcpyDeviceToHost));
  gpuCheck(hipFree(d_a));
  gpuCheck(hipFree(d_b));
  gpuCheck(hipFree(d_c));  
}

} // unnamed namespace

namespace wrapper {

void matMulV1(const std::vector<float>& a,
              const std::vector<float>& b,
              std::vector<float>& c,              
              int i,
              int j,
              int k) {
  kernelRunner(a, b, c, i, j, k, ::matmulV1);
}

// TODO: hook it up with the kernel that uses shared memory
void matMulV2(const std::vector<float>& a,
              const std::vector<float>& b,
              std::vector<float>& c,              
              int i,
              int j,
              int k) {
  kernelRunner(a, b, c, i, j, k, ::matmulV1);
}

void print_cuda_properties() {
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  fprintf(stdout, "Warp size: %d\n", props.warpSize);
  fprintf(stdout, "Max grid size: %d, %d, %d\n", props.maxGridSize[0], props.maxGridSize[1], props.maxGridSize[2]);  
}
}
