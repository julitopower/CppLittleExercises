#include "hip/hip_runtime.h"
#include <cstdio>

#include "hip/hip_runtime.h"


#include "main.cuh"

#define gpuCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void fn(int n, const float* v1, const float* v2, float* out) {
  // Each kernel call processes one element
  // idx = blockIdx.x * blockDim.x + threadIdx
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n) {
    return;
  }
  out[idx] = v1[idx] + v2[idx];
}

namespace wrapper {
void wrapper(const std::vector<float>& v1,
             const std::vector<float>& v2,
             std::vector<float>& out) {
  const auto n = v1.size();
  float *d_v1, *d_v2, *d_out;

  gpuCheck(hipMalloc(&d_v1, sizeof(float) * n));
  gpuCheck(hipMalloc(&d_v2, sizeof(float) * n));
  gpuCheck(hipMalloc(&d_out, sizeof(float) * n));
  gpuCheck(hipMemcpy(d_v1, v1.data(), sizeof(float) * n, hipMemcpyHostToDevice));
  gpuCheck(hipMemcpy(d_v2, v2.data(), sizeof(float) * n, hipMemcpyHostToDevice));
  const auto threads = 32;
  const auto blocks = (n + threads - 1) / threads;
  fn<<<blocks,threads>>>(v1.size(), d_v1, d_v2, d_out);
  hipDeviceSynchronize();
  
  gpuCheck(hipMemcpy(out.data(), d_out, sizeof(float) * n, hipMemcpyDeviceToHost));
  gpuCheck(hipFree(d_v1));
  gpuCheck(hipFree(d_v2));
  gpuCheck(hipFree(d_out));
}
}